#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 16

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}

void matrix_mul(int *a,int *b, int *c){
	int *tempA;
	int *tempB;
	int *tempC;
	for (int i = 0 ; i < 4 ; i++ ){
		for (int k = 0 ; k < 4 ; k++ ){
			int temporal = 0 ;
			for (int j = 0 ; j < 4 ; j++ ){
				tempA = a+(4*i+j);
				tempB = b+(4*j+k);
				tempC = c+(4*i+k);
				temporal += (*tempA)*(*tempB);
				*tempC = temporal;
			}
		}
	}
}

void Filling_Matrix(int *a){
	int *temp = a;
	srand(time(NULL));
	for(int i=0;i<17;i++){
		*temp = rand();
		temp++;
	}
}

__global__ void matrix_multiplication( int *a, int *b, int *c ) {

	int Filas = blockIdx.y*blockDim.y+threadIdx.y;
	int Columnas = blockIdx.x*blockDim.x+threadIdx.x;

	float SumaTemporal = 0;

	if (Filas < N && Columnas < N) {// cada thread se encarga de un bloque de la sub matrix
		for (int i = 0; i < N; i++) {
			SumaTemporal += a[Filas * N + i] * b[i * N + Columnas];
		}
	}
	c[Filas * N + Columnas] = SumaTemporal;
}

int main( void ) {
	
	clock_t t_ini, t_fin;	

	int *a, *b, *c;           // datos en el host    
	int *dev_a, *dev_b, *dev_c;   //datos en el dispositivo
	int size = N * sizeof( int ); // asignación de memoria
	
	// asginación de memoria para los dispositivos
	hipMalloc( (void**)&dev_a, size );
	hipMalloc( (void**)&dev_b, size );
	hipMalloc( (void**)&dev_c, size );
	
	a = (int*)malloc( size );
	b = (int*)malloc( size );
	c = (int*)malloc( size );
	
	//se encarga de llenar de datos las matrices
	Filling_Matrix(a);
	Filling_Matrix(b);
		
	// entradas copiadas a los dispotivos
	hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

//se ejecuta la multiplicación con cuda
	t_ini = clock();
	matrix_multiplication<<<1,16>>>(dev_a,dev_b,dev_c);
	t_fin = clock();
	// copy device result back to host copy of c
	hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost );
	
	printf("CUDA TIME %f \n \n",(double)(t_fin - t_ini));

//Calculo  sin cuda
        t_ini = clock();
        matrix_mul(a,b,c);
        t_fin = clock();
        printf("CPU TIME %f \n \n",(double)(t_fin - t_ini));
	
	free( a );
	free( b );
	free( c );
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );
	
	return 0;
}

